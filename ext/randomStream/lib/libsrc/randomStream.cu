#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "randomStream.h"
//#include <hip/hip_runtime.h>
extern "C" {

  __global__ void CudaPCG( uint64_t *state, uint64_t *inc , double* output){
    uint64_t oldstate = state[threadIdx.x + blockIdx.x * blockDim.x];
    // Advance internal state
    state[threadIdx.x + blockIdx.x * blockDim.x] = oldstate * 6364136223846793005ULL + (inc[threadIdx.x + blockIdx.x * blockDim.x]|1);
    // Calculate output function (XSH RR), uses old state for max ILP
    //uint32_t xorshifted = (((oldstate >> 18u) ^ oldstate) >> 27u);
    //uint32_t rot = (oldstate >> 59u);
    uint32_t buff = ((((oldstate >> 18u) ^ oldstate) >> 27u) >> (oldstate >> 59u)) | ((((oldstate >> 18u) ^ oldstate) >> 27u) << ((-(oldstate >> 59u)) & 31));
    output[threadIdx.x + blockIdx.x * blockDim.x] = (double)buff/(double)UINT32_MAX;
  }

  __global__ void CudaInitializeStates( uint64_t* state ){
    state[threadIdx.x + blockIdx.x * blockDim.x] = threadIdx.x + blockIdx.x * blockDim.x;
  }


  void reCalculateStream( randomStream* stream ){
    hipMemcpy(stream->randoms, stream->devOutput, stream->size * sizeof(double), hipMemcpyDeviceToHost);
    CudaPCG<<<stream->size/threads_per_block,threads_per_block>>>( stream->stateArray, stream->incArray, stream->devOutput );
    stream->used = 0;
  }

  double getRandom( randomStream* stream ){
    double toret;

    if (stream->size == stream->used +1 ){
      reCalculateStream( stream );
    }

    toret = stream->randoms[stream->used];
    stream->used++;
    return toret;
  }

  randomStream* createRandomStream( int size ){
    for(; size%threads_per_block != 0; size++);

    randomStream *stream = (randomStream*) malloc(sizeof(randomStream));
    stream->randoms = (double *)malloc(size*sizeof(double));
    stream->size = size;

    hipMalloc((void **)&stream->stateArray, stream->size* sizeof(uint64_t));
    hipMalloc((void **)&stream->devOutput, stream->size* sizeof(double));
    hipMalloc((void **)&stream->incArray, stream->size* sizeof(uint64_t));

    CudaInitializeStates<<<stream->size/threads_per_block,threads_per_block>>>( stream->stateArray );
    CudaPCG<<<stream->size/threads_per_block,threads_per_block>>>( stream->stateArray, stream->incArray, stream->devOutput );
    CudaPCG<<<stream->size/threads_per_block,threads_per_block>>>( stream->stateArray, stream->incArray, stream->devOutput );
    reCalculateStream( stream );

    return stream;
  }


  void destroyStream( randomStream* stream ){
    free( stream->randoms );
    hipFree( stream->stateArray );
    hipFree( stream->incArray );
    hipFree( stream->devOutput );
    free( stream );
  }
}
